#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/core/TensorBase.h>
#include <ATen/OpMathType.h>
#include <ATen/Dispatch.h>

#include <ATen/native/cuda/ScanKernels.h>
#include <ATen/native/cuda/ScanUtils.cuh>

#include <cmath>
#include <limits>

namespace at::native {

void launch_logcumsumexp_cuda_kernel(const TensorBase& result, const TensorBase& self, int64_t dim) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      self.scalar_type(), "logcumsumexp_cuda",
      [&]() {
        using opmath_t = at::opmath_type<scalar_t>;
        scalar_t init = -std::numeric_limits<scalar_t>::infinity();
        auto log_add_exp = [] C10_HOST_DEVICE (const scalar_t x_, const scalar_t y_) -> scalar_t {
          const opmath_t x{x_}, y{y_};
          auto min = at::_isnan(y) ? y : std::min<opmath_t>(x, y); //std::min returns first arg if one of the args is nan
          auto max = at::_isnan(y) ? y : std::max<opmath_t>(x, y); //std::max returns first arg if one of the args is nan
          if (min != max || ::isfinite(min)) {
          // nan will be propagated here
              return ::log1p(std::exp(min - max)) + max;
          } else {
          // special case to correctly handle infinite inputs
             return x;
          }
        };
        scan_dim<scalar_t>(self, result, dim, init, log_add_exp);
      });
}

} // namespace at::native
